#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <iostream>
#include <chrono>
#include <math.h>

using namespace std;


struct functor 
{
   int mov_count;
   int n_cat;
   int *lim_cats;
   movie *movies;

   functor(int _mov_count, movie *_movies, int _n_cat ,int *_lim_cats) : mov_count(_mov_count), movies(_movies), n_cat(_n_cat) ,lim_cats(_lim_cats) {
      
   };
   __device__ __host__
   int operator() (int selection_it)
   {  

      bool slots[30];
      for(int h = 0; h < 30; h++) {
        slots[h] = false;
      }

      int lim_cats_cp[30];
      for(int h = 0; h <= n_cat; h++) {
        lim_cats_cp[h] = lim_cats[h];
      }

      int added = 0;
      for(int i = 0; i < mov_count; i++)
      {
          if(added > 24) return -1;
          if(selection_it & (1 << i))
          {
              if(movies[i].end == movies[i].start) {
                if(slots[movies[i].start]) return -1;
                slots[movies[i].start] = true;
              } 
              else {
                for(int j = movies[i].start; j < movies[i].end; j++) {
                  if(slots[j]) return -1;
                  slots[j] = true;
                }
              }
              //cout << movies[i].cat << endl;
              if(lim_cats_cp[movies[i].cat] == 0) return -1;
              lim_cats_cp[movies[i].cat] --;
              added++;
          }
      }
      return added;
   }
};

void test_combinations(vector<movie> &movies, vector<int> &lim_cats, int n_cat) {
  chrono::steady_clock::time_point begin = get_time();
  
  thrust::device_vector<int> cats_lim_gpu(lim_cats);
  //cats_lim_gpu = lim_cats;

  thrust::device_vector<movie> mov_gpu(movies);
  //mov_gpu = movies;

  thrust::device_vector<int> mov_count(pow(2, movies.size()));
  
  thrust::counting_iterator<int> comb(0);

  thrust::transform(comb, comb + pow(2, movies.size()), mov_count.begin(), functor(movies.size(), thrust::raw_pointer_cast(mov_gpu.data()), n_cat ,thrust::raw_pointer_cast(cats_lim_gpu.data())));
  int best = *thrust::max_element(mov_count.begin(), mov_count.end());

  chrono::steady_clock::time_point end = get_time();

  cout << get_interval(begin, end) << 'x' << pow(2, movies.size()) << 'x' << pow(2, movies.size());
}

int main(int argc, char *argv[]) {
  int n_mov, n_cat;

  vector<movie> movies, selected;
  vector<int> lim_cats;

  cin >> n_mov >> n_cat;
  
  movies.reserve(n_mov);

  read_cats_limit(lim_cats, n_cat);
  read_movies_data(movies, n_mov);

  test_combinations(movies, lim_cats, n_cat);

}